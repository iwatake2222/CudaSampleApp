#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "CudaLib.h"

namespace CudaLib
{
#if 0
}	// indent guard
#endif


void complexCalcOriginal(int *in, int *out, int n)
{
	for (int i = 0; i < n; i++) {
		out[i] = in[i] * 2;
	}
}


__global__ void complexCalcFastLoop(int *in, int *out, int n)
{
	int i = threadIdx.x;
	if (i < n) {
		out[i] = in[i] * 2;
	}
}

void complexCalcFast(int *hIn, int *hOut, int n)
{
	int *dIn;
	int *dOut;
	hipHostMalloc((void**)&dIn, n * sizeof(int));
	hipHostMalloc((void**)&dOut, n * sizeof(int));
	hipMemcpy(dIn, hIn, n * sizeof(int), hipMemcpyHostToDevice);

	complexCalcFastLoop <<<1, n>>> (dIn, dOut, n);
	hipDeviceSynchronize();

	hipMemcpy(hOut, dOut, n * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dIn);
	hipFree(dOut);
}

}
