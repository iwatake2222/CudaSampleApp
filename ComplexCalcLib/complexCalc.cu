#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include "stdlib.h"

#include "complexCalc.h"

namespace CudaLib
{
#if 0
}	// indent guard
#endif

void complexCalcOriginal(int n)
{
	for (int i = 0; i < n; i++) {
		printf("processing heavy work (%d / %d)\n", i, n);
	}
}

void complexCalc2Original(int *in, int*out, int n)
{
	for (int i = 0; i < n; i++) {
		out[i] = in[i] * 2;
	}
}

void allocManaged(int **p, int size)
{
	hipMallocManaged(p, size);
}

void freeManaged(int *p)
{
	hipFree(p);
}

__global__ void complexCalcFastLoop(int n)
{
	int i = threadIdx.x;
	if (i < n) {
		printf("processing heavy work (%d / %d)\n", i, n);
	}
}

void complexCalcFast(int n)
{
	complexCalcFastLoop << <1, n >> > (n);
	hipDeviceSynchronize();
}

__global__ void complexCalc2FastLoop(int *in, int*out, int n)
{
	int i = threadIdx.x;
	if (i < n) {
		out[i] = in[i] * 2;
	}
}

void complexCalc2Fast(int *in, int*out, int n)
{
	complexCalc2FastLoop << <1, n >> > (in, out, n);
	hipDeviceSynchronize();
}

}